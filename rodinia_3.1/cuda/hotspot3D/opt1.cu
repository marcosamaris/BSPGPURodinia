#include "hip/hip_runtime.h"
__global__ void hotspotOpt1(float *p, float* tIn, float *tOut, float sdc,
        int nx, int ny, int nz,
        float ce, float cw, 
        float cn, float cs,
        float ct, float cb, 
        float cc) 
{
    float amb_temp = 80.0;

    int i = blockDim.x * blockIdx.x + threadIdx.x;  
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    int c = i + j * nx;
    int xy = nx * ny;

    int W = (i == 0)        ? c : c - 1;
    int E = (i == nx-1)     ? c : c + 1;
    int N = (j == 0)        ? c : c - nx;
    int S = (j == ny-1)     ? c : c + nx;

    float temp1, temp2, temp3;
    temp1 = temp2 = tIn[c];
    temp3 = tIn[c+xy];
    tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
        + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
    c += xy;
    W += xy;
    E += xy;
    N += xy;
    S += xy;

    for (int k = 1; k < nz-1; ++k) {
        temp1 = temp2;
        temp2 = temp3;
        temp3 = tIn[c+xy];
        tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
            + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
        c += xy;
        W += xy;
        E += xy;
        N += xy;
        S += xy;
    }
    temp1 = temp2;
    temp2 = temp3;
    tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
        + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
    return;
}

void hotspot_opt1(float *p, float *tIn, float *tOut,
        int nx, int ny, int nz,
        float Cap, 
        float Rx, float Ry, float Rz, 
        float dt, int numiter) 
{

    uint64_t time1=0, time2=0, totalTime=0, totalTime2;
    
    float ce, cw, cn, cs, ct, cb, cc;
    float stepDivCap = dt / Cap;
    ce = cw =stepDivCap/ Rx;
    cn = cs =stepDivCap/ Ry;
    ct = cb =stepDivCap/ Rz;

    cc = 1.0 - (2.0*ce + 2.0*cn + 3.0*ct);

    size_t s = sizeof(float) * nx * ny * nz;  
    float  *tIn_d, *tOut_d, *p_d;
    hipMalloc((void**)&p_d,s);
    hipMalloc((void**)&tIn_d,s);
    hipMalloc((void**)&tOut_d,s);
    hipMemcpy(tIn_d, tIn, s, hipMemcpyHostToDevice);
    hipMemcpy(p_d, p, s, hipMemcpyHostToDevice);

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(hotspotOpt1), hipFuncCachePreferL1);

    dim3 block_dim(64, 4, 1);
    dim3 grid_dim(nx / 64, ny / 4, 1);

    totalTime1 = getTime();
    for (int i = 0; i < numiter; ++i) {
        time1 = getTime();
        hotspotOpt1<<<grid_dim, block_dim>>>
            (p_d, tIn_d, tOut_d, stepDivCap, nx, ny, nz, ce, cw, cn, cs, ct, cb, cc);
        time2 = getTime();
        printf("1, hotspot3D, %d, %d, %d, %d, %d, %d, \n", numiter, i, nx, ny, nz, (uint64_t)(time2 - time1));
        
        float *t = tIn_d;
        tIn_d = tOut_d;
        tOut_d = t;
    }
    totalTime2 = getTime();
    printf("1, , , , , , , %d\n",  (uint64_t)(totalTime2 - totalTime1));

    hipDeviceSynchronize();
    hipMemcpy(tOut, tOut_d, s, hipMemcpyDeviceToHost);
    hipFree(p_d);
    hipFree(tIn_d);
    hipFree(tOut_d);
    return;
}

